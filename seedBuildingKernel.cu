#include "hip/hip_runtime.h"
// Kernel that does the seed building / 
// Everything should be done in here since kernel launches are costly
// Maybe use device only kernels for functions?

/* Info to be stored in SoA seed struct */
//nhits
// Collection of rechits per seed
//rechits.isValid
//rechits.globalPosition
//rechits.geographicalID

__global__  void processSeed(seedInfo, ecalSCinfo,trajState,matchedSeeds*){


// Easy to pass  - in the Ecal SC struct
eta =  ecalSCinfo.eta
et = ecalSCinfo.energy * sin(ecalSCinfo.theta)

// this is 1 or -1
trajState.charge

// Parameter from config - should be passed as kernel argument
const auto nCuts = cfg_.matchingCuts.size(); 
const auto enableHitSkipping  = cfg_.enableHitSkipping
//These I think are the same for all seeds 
//////


//=== Loop over seeds 

// Create a vector of stucts or cuda friendly object to store matches
matches [1]
//Free trajStare object
firstMatchFreeTraj
//And two Global point structures
prevHitPosition
vertex 


//*** Loop over seed hits
for (size_t iHit = 0;matches.size() < nCuts && iHit < seed.nHits() && (cfg_.enableHitSkipping || iHit == matches.size());iHit++) {

    //access rechit --> Should be part of seed struct
    auto const& recHit = *(seed.recHits().begin() + iHit);

    //check if rechit is valid
    if (!recHit.isValid())
        continue;
    // Add var that check if this is the first attempt to perfrom a matching
    const bool doFirstMatch = matches.empty();

    // Check if the first match should be made 
    auto const& trajState = doFirstMatch? getTrajStateFromVtx(recHit, initialTrajState, backwardPropagator_)  : getTrajStateFromPoint(recHit, firstMatchFreeTraj, prevHitPos, forwardPropagator_);

    // If true -> getTrajStateFromVtx
    // If false -> getTrajStateFromPoint
    // 
    
    *Propagator* https://cmssdt.cern.ch/dxr/CMSSW/source/TrackingTools/MaterialEffects/interface/PropagatorWithMaterial.h#25
 https://cmssdt.cern.ch/dxr/CMSSW/source/RecoEgamma/EgammaElectronAlgos/src/TrajSeedMatcher.cc#115
    enum PropagationDirection { oppositeToMomentum, alongMomentum, anyDirection, invalidDirection };
    https://cmssdt.cern.ch/dxr/CMSSW/source/MagneticField/Engine/interface/MagneticField.h#19
    https://cmssdt.cern.ch/dxr/CMSSW/source/TrackingTools/MaterialEffects/src/PropagatorWithMaterial.cc





//***


//===


}


From initial trajectory :
trajState(ecalSCinfo,charge)

charge 



[1] 
  struct SCHitMatch {

    const DetId detId = 0;

    const GlobalPoint hitPos; <-- this we can make a struct 

    const float dRZ = std::numeric_limits<float>::max();
    const float dPhi = std::numeric_limits<float>::max();

    const TrackingRecHit& hit; <--- Check this

    const float et = 0.f;
    const float eta = 0.f;
    const float phi = 0.f;
    const int charge = 0;
    const int nrClus = 0;
  };

