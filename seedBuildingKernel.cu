#include "hip/hip_runtime.h"
// Kernel that does the seed building / 
// Everything should be done in here since kernel launches are costly
// Maybe use device only kernels for functions?

/* Info to be stored in SoA seed struct */
//nhits
// Collection of rechits per seed
//rechits.isValid
//rechits.globalPosition
//rechits.geographicalID

__global__  void processSeed(seedInfo, ecalSCinfo,trajState,matchedSeeds*){


// Easy to pass  - in the Ecal SC struct
eta =  ecalSCinfo.eta
et = ecalSCinfo.energy * sin(ecalSCinfo.theta)

// this is 1 or -1
trajState.charge

// Parameter from config - should be passed as kernel argument
const auto nCuts = cfg_.matchingCuts.size(); 
const auto enableHitSkipping  = cfg_.enableHitSkipping
//These I think are the same for all seeds 
//////


//=== Loop over seeds 

// Create a vector of stucts or cuda friendly object to store matches
matches [1]
//Free trajStare object
firstMatchFreeTraj
//And two Global point structures
prevHitPosition
vertex 


//Need to replace
/*
TsosWP with cuda compatable data structure I think tsos will be replaced by a data structure  
*/

//Information needed for PropogateWithPath
/*
Class Inputs: propagation direction, mass and magnetic field are sepcified
Defualt parameters: maxDphi = 1.6, useRungeKutta = false, ptMin = -1, useOldGeoPropLogic = true)
Objects created by the class: AnalyticalPropagator()
Arguements for propagate with path: fts and surface (either plane or cylinder) these come from the initial state and the surface type of hit.det()


*/

//AnalyticalPropagator
/*
Class inputs: Magnetic field (passed in from MaterialPropagator), PropagationDir dir (Also passed in from MaterailPropagator), maxDPhi = 1.6, isOld = True.
PropogateWithPath Method inputs: fts, and surface (either cylinder or plane)


*/

//*** Loop over seed hits
for (size_t iHit = 0;matches.size() < nCuts && iHit < seed.nHits() && (cfg_.enableHitSkipping || iHit == matches.size());iHit++) {

    //access rechit --> Should be part of seed struct
    auto const& recHit = *(seed.recHits().begin() + iHit);

    //check if rechit is valid
    if (!recHit.isValid())
        continue;
    // Add var that check if this is the first attempt to perfrom a matching
    const bool doFirstMatch = matches.empty();

    // Check if the first match should be made 
    auto const& trajState = doFirstMatch? getTrajStateFromVtx(recHit, initialTrajState, backwardPropagator_)  : getTrajStateFromPoint(recHit, firstMatchFreeTraj, prevHitPos, forwardPropagator_);

    // If true -> getTrajStateFromVtx
    // If false -> getTrajStateFromPoint
    
    //vtx vs point check (difference between bacwardPropagator_ and forwardPropagator_)
    if (doFirstMatch){
      auto propDir = oppositeToMomentum;
    }else{
      auto propDir = alongMomentum;
    }
    
    //Attempt to reproduce PropagatorWithMaterial::propagateWithPath methods
    //
    // add material at starting surface, if requested
    //
    TsosWP newTsosWP(tsos, 0.);
    if (materialAtSource()) {
      bool updateOk = theMEUpdator->updateStateInPlace(newTsosWP.first, propDir); //MEupdator.updateStateInPlace needs to be implemented
      // MEUpdator functionality reproduced
       
      if UNLIKELY (!updateOk)
        newTsosWP.first = TrajectoryStateOnSurface();
    }
    if UNLIKELY (!newTsosWP.first.isValid())
      return newTsosWP; //Break?
    //
    // geometrical propagation (Analytic propagators)
    //
    if (dynamic_cast<Cylinder*>(recHit.det()->surface()) == nullptr){ //Planar Geometry
      // check curvature
      float rho = fts.transverseCurvature();
      // propagate parameters
      GlobalPoint x;
      GlobalVector p;
      double s;
      // check if already on plane
      if LIKELY (plane.localZclamped(fts.position()) != 0) {
        // propagate
        bool parametersOK = this->propagateParametersOnPlane(fts, plane, x, p, s); //This method needs to be implemented
        // check status and deltaPhi limit
        float dphi2 = float(s) * rho;
        dphi2 = dphi2 * dphi2 * fts.momentum().perp2();
        if UNLIKELY (!parametersOK || dphi2 > theMaxDPhi2 * fts.momentum().mag2())
          newTsosWP = TsosWP(TrajectoryStateOnSurface(), 0.);
        } else {
          LogDebug("AnalyticalPropagator") << "not going anywhere. Already on surface.\n"
                                           << "plane.localZ(fts.position()): " << plane.localZ(fts.position()) << "\n"
                                           << "plane.position().mag(): " << plane.position().mag() << "\n"
                                           << "plane.posPrec: " << plane.posPrec(); //Is the log statement realy needed?

          x = fts.position();
          p = fts.momentum();
          s = 0;
        }
        //
        // Compute propagated state and check change in curvature
        //
        GlobalTrajectoryParameters gtp(x, p, fts.charge(), theField);
        if UNLIKELY (std::abs(gtp.transverseCurvature() - rho) > theMaxDBzRatio * std::abs(rho))
          newTsosWP = TsosWP(TrajectoryStateOnSurface(), 0.);
        newTsosWP = propagatedStateWithPath(fts, plane, gtp, s);
    }
    if (dynamic_cast<Plane*>(recHit.det()->surface()) == nullptr){ //Cylindrical Geometry
      // check curvature
      auto rho = fts.transverseCurvature();
      // propagate parameters
      GlobalPoint x;
      GlobalVector p;
      double s = 0;
      bool parametersOK = this->propagateParametersOnCylinder(fts, cylinder, x, p, s); //This method needs to be implemented
      // check status and deltaPhi limit
      float dphi2 = s * rho;
      dphi2 = dphi2 * dphi2 * fts.momentum().perp2();

      if UNLIKELY (!parametersOK || dphi2 > theMaxDPhi2 * fts.momentum().mag2())
        newTsosWP = TsosWP(TrajectoryStateOnSurface(), 0.);

      GlobalTrajectoryParameters gtp(x, p, fts.charge(), theField);

      if UNLIKELY (std::abs(gtp.transverseCurvature() - rho) > theMaxDBzRatio * std::abs(rho))
        newTsosWP = TsosWP(TrajectoryStateOnSurface(), 0.);

      ConstReferenceCountingPointer<TangentPlane> plane(
          cylinder.tangentPlane(x));  // need to be here until tsos is created!
      newTsosWP = propagatedStateWithPath(fts, *plane, gtp, s); //This needs to be implemented
      }

      if UNLIKELY (!(newTsosWP.first).isValid() || materialAtSource())
        return newTsosWP;
      //
      // add material at destination surface, if requested
      //
      bool updateOk = theMEUpdator->updateStateInPlace(
          newTsosWP.first, PropagationDirectionFromPath()(newTsosWP.second, propDir));
      if UNLIKELY (!updateOk)
        newTsosWP.first = TrajectoryStateOnSurface(); 
      return newTsosWP;

    


 
    *Propagator* https://cmssdt.cern.ch/dxr/CMSSW/source/TrackingTools/MaterialEffects/interface/PropagatorWithMaterial.h#25
 https://cmssdt.cern.ch/dxr/CMSSW/source/RecoEgamma/EgammaElectronAlgos/src/TrajSeedMatcher.cc#115
    enum PropagationDirection { oppositeToMomentum, alongMomentum, anyDirection, invalidDirection };
    https://cmssdt.cern.ch/dxr/CMSSW/source/MagneticField/Engine/interface/MagneticField.h#19
    https://cmssdt.cern.ch/dxr/CMSSW/source/TrackingTools/MaterialEffects/src/PropagatorWithMaterial.cc





//***


//===


}


From initial trajectory :
trajState(ecalSCinfo,charge)

charge 



[1] 
  struct SCHitMatch {

    const DetId detId = 0;

    const GlobalPoint hitPos; <-- this we can make a struct 

    const float dRZ = std::numeric_limits<float>::max();
    const float dPhi = std::numeric_limits<float>::max();

    const TrackingRecHit& hit; <--- Check this

    const float et = 0.f;
    const float eta = 0.f;
    const float phi = 0.f;
    const int charge = 0;
    const int nrClus = 0;
  };




  std::pair<TrajectoryStateOnSurface, double> AnalyticalPropagator::propagatedStateWithPath(
        const FreeTrajectoryState& fts,
        const Surface& surface,
        const GlobalTrajectoryParameters& gtp,
        const double& s) const {
      //
      // for forward propagation: state is before surface,
      // for backward propagation: state is after surface
      //
      SurfaceSide side =
          PropagationDirectionFromPath()(s, propagationDirection()) == alongMomentum ? beforeSurface : afterSurface;
      //
      //
      // error propagation (if needed) and conversion to a TrajectoryStateOnSurface
      //
      if (fts.hasError()) {
        //
        // compute jacobian
        //
        AnalyticalCurvilinearJacobian analyticalJacobian(fts.parameters(), gtp.position(), gtp.momentum(), s);
        const AlgebraicMatrix55& jacobian = analyticalJacobian.jacobian();
        // CurvilinearTrajectoryError cte(ROOT::Math::Similarity(jacobian, fts.curvilinearError().matrix()));
        return TsosWP(
            TrajectoryStateOnSurface(gtp, ROOT::Math::Similarity(jacobian, fts.curvilinearError().matrix()), surface, side),
            s);
      } else {
        //
        // return state without errors
        //
        return TsosWP(TrajectoryStateOnSurface(gtp, surface, side), s);
      }
    }

